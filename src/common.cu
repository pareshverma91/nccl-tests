#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2016-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENCE.txt for license information
 ************************************************************************/

#include "common.h"
#include <pthread.h>
#include <cstdio>
#include <getopt.h>
#include "hip/hip_runtime.h"

#if NCCL_MAJOR >= 2
ncclDataType_t test_types[ncclNumTypes] = {ncclInt8, ncclUint8, ncclInt32, ncclUint32, ncclInt64, ncclUint64, ncclHalf, ncclFloat, ncclDouble};
const char *test_typenames[ncclNumTypes] = {"int8", "uint8", "int32", "uint32", "int64", "uint64", "half", "float", "double"};
#else
ncclDataType_t test_types[ncclNumTypes] = {ncclChar, ncclInt, ncclHalf, ncclFloat, ncclDouble, ncclInt64, ncclUint64};
const char *test_typenames[ncclNumTypes] = {"char", "int", "half", "float", "double", "int64", "uint64"};
#endif
ncclRedOp_t test_ops[ncclNumOps] = {ncclSum, ncclProd, ncclMax, ncclMin};
const char *test_opnames[ncclNumOps] = {"sum", "prod", "max", "min"};

thread_local int is_main_thread = 0;

static int datacheck = 1;
static int warmup_iters = 5;
static int iters = 20;
static int ncclop = ncclSum;
static int nccltype = ncclFloat;
static int ncclroot = 0;
static int swap_args = 0;
static int parallel_init = 0;
static int blocking_coll = 0;
char *ncclhost = NULL;

double parsesize(char *value) {
    long long int units;
    double size;

    if (strchr(value, 'G') != NULL) {
        units=1024*1024*1024;
    } else if (strchr(value, 'M') != NULL) {
        units=1024*1024;
    } else if (strchr(value, 'K') != NULL) {
        units=1024;
    } else {
        units=1;
    }

    size = atof(value)*units;
    return size;
}

double DeltaMaxValue(ncclDataType_t type) {
  switch(type) {
    case ncclHalf: return 1e-2;
    case ncclFloat: return 1e-5;
    case ncclDouble: return 1e-12;
    case ncclInt:
#if NCCL_MAJOR >= 2
    case ncclUint8:
    //case ncclInt32:
    case ncclUint32:
#endif
    case ncclInt64:
    case ncclUint64: return 1e-200;
  }
  return 1e-200;
}

template<typename T> __device__
double absDiff(T a, T b) {
  return fabs((double)(b - a));
}

template<> __device__
double absDiff<half>(half a, half b) {
  float x = __half2float(a);
  float y = __half2float(b);
  return fabs((double)(y-x));
}

template<typename T> __device__
float toFloat(T a) {
  return (float)a;
}
template<> __device__
float toFloat(half a) {
  return __half2float(a);
}


template<typename T, int BSIZE> __global__
void deltaKern(void* A_, void* B_, size_t count, double* max) {
  const T* A = (const T*)A_;
  const T* B = (const T*)B_;
  __shared__ double temp[BSIZE];
  int tid = threadIdx.x;
  double locmax = 0.0;
  for(int i=tid; i<count; i+=blockDim.x) {

    double delta = absDiff(A[i], B[i]);
    if( delta > locmax ) {
      locmax = delta;
#ifdef DEBUG_PRINT
      if (delta > .1) printf("Error at %d/%d : %f != %f\n", i, count, toFloat(A[i]), toFloat(B[i]));
#endif
    }
  }

  temp[tid] = locmax;
  for(int stride = BSIZE/2; stride > 1; stride>>=1) {
    __syncthreads();
    if( tid < stride )
      temp[tid] = temp[tid] > temp[tid+stride] ? temp[tid] : temp[tid+stride];
  }
  __syncthreads();
  if( threadIdx.x == 0)
    *max = temp[0] > temp[1] ? temp[0] : temp[1];
}


void CheckDelta(void* expected, void* results, size_t count, ncclDataType_t type, double* devmax) {
  switch (type) {
    case ncclHalf:
      deltaKern<half, 512><<<1, 512>>>(results, expected, count, devmax); break;
    case ncclFloat:
      deltaKern<float, 512><<<1, 512>>>(results, expected, count, devmax); break;
    case ncclDouble:
      deltaKern<double, 512><<<1, 512>>>(results, expected, count, devmax); break;

    case ncclChar:
#if NCCL_MAJOR >= 2
    case ncclUint8:
#endif
      deltaKern<uint8_t, 512><<<1, 512>>>(results, expected, count, devmax); break;
    case ncclInt:
#if NCCL_MAJOR >= 2
    case ncclUint32:
#endif
      deltaKern<uint32_t, 512><<<1, 512>>>(results, expected, count, devmax); break;
    case ncclInt64:
    case ncclUint64:
      deltaKern<uint64_t, 512><<<1, 512>>>(results, expected, count, devmax); break;
  }
}

#define CURAND_CHK(cmd)                                                         \
    do {                                                                        \
      hiprandStatus_t error = (cmd);                                             \
      if (error != HIPRAND_STATUS_SUCCESS) {                                     \
        printf("CuRAND error %i at %s:%i\n", error, __FILE__ , __LINE__);       \
        exit(EXIT_FAILURE);                                                     \
      }                                                                         \
    } while (false)


template<typename T>
void GenerateRandom(hiprandGenerator_t generator, T * const dest,
    const size_t N);

template<>
void GenerateRandom<int8_t>(hiprandGenerator_t generator, int8_t * const dest,
    const size_t N) {
  size_t align = (4 - (((size_t)dest) & 3)) % 4;
  CURAND_CHK(hiprandGenerate(generator, (unsigned int*)(dest+align),
      N * sizeof(int8_t) / sizeof(int)));
  CUDACHECK(hipMemcpy(dest, dest+4, align, hipMemcpyDeviceToDevice));
}
template<>
void GenerateRandom<uint8_t>(hiprandGenerator_t generator, uint8_t * const dest,
    const size_t N) {
  size_t align = (4 - (((size_t)dest) & 3)) % 4;
  CURAND_CHK(hiprandGenerate(generator, (unsigned int*)(dest+align),
      N * sizeof(uint8_t) / sizeof(int)));
  CUDACHECK(hipMemcpy(dest, dest+4, align, hipMemcpyDeviceToDevice));
}

template<>
void GenerateRandom<int32_t>(hiprandGenerator_t generator, int32_t * const dest,
    const size_t N) {
  CURAND_CHK(hiprandGenerate(generator, (unsigned int*)dest, N));
}

template<>
void GenerateRandom<uint32_t>(hiprandGenerator_t generator, uint32_t * const dest,
    const size_t N) {
  CURAND_CHK(hiprandGenerate(generator, (unsigned int*)dest, N));
}

template<>
void GenerateRandom<float>(hiprandGenerator_t generator, float * const dest,
    const size_t N) {
  CURAND_CHK(hiprandGenerateUniform(generator, dest, N));
}

template<>
void GenerateRandom<double>(hiprandGenerator_t generator, double * const dest,
    const size_t N) {
  CURAND_CHK(hiprandGenerateUniformDouble(generator, dest, N));
}

template<>
void GenerateRandom<uint64_t>(hiprandGenerator_t generator, uint64_t * const dest,
    const size_t N) {
  CURAND_CHK(hiprandGenerate(generator, (unsigned int *)dest, N*2));
}

template<>
void GenerateRandom<int64_t>(hiprandGenerator_t generator, int64_t * const dest,
    const size_t N) {
  CURAND_CHK(hiprandGenerate(generator, (unsigned int *)dest, N*2));
}

template<typename T>
void RandomizeType(void* dest, const size_t N, const int randomSeed) {
  T* ptr = (T*)dest;
  hiprandGenerator_t gen;
  CURAND_CHK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32));
  CURAND_CHK(hiprandSetPseudoRandomGeneratorSeed(gen, randomSeed));
  GenerateRandom<T>(gen, ptr, N);
  CURAND_CHK(hiprandDestroyGenerator(gen));
  CUDACHECK(hipDeviceSynchronize());
}

__global__ void halve(const float * src, half* dest, size_t N) {
  for(int tid = threadIdx.x + blockIdx.x*blockDim.x;
      tid < N; tid += blockDim.x * gridDim.x)
    dest[tid] = __float2half(src[tid]);
}

void RandomizeHalf(void* dest, const size_t N, const int randomSeed) {
  half* ptr = (half*)dest;
  hiprandGenerator_t gen;
  CURAND_CHK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32));
  CURAND_CHK(hiprandSetPseudoRandomGeneratorSeed(gen, randomSeed));

  float* temp;
  CUDACHECK(hipMalloc(&temp, N*sizeof(float)));
  GenerateRandom<float>(gen, temp, N);
  halve<<<128, 512>>>(temp, ptr, N);
  CURAND_CHK(hiprandDestroyGenerator(gen));
  CUDACHECK(hipFree(temp));
  CUDACHECK(hipDeviceSynchronize());
}

void Randomize(void* ptr, const size_t count, ncclDataType_t type, const int seed) {
  switch (type) {
    case ncclChar:   RandomizeType<int8_t>  (ptr, count, seed); break;
#if NCCL_MAJOR >= 2
    case ncclUint8:  RandomizeType<uint8_t> (ptr, count, seed); break;
#endif
    case ncclInt:    RandomizeType<int32_t> (ptr, count, seed); break;
#if NCCL_MAJOR >= 2
    case ncclUint32: RandomizeType<uint32_t>(ptr, count, seed); break;
#endif
    case ncclInt64:  RandomizeType<int64_t> (ptr, count, seed); break;
    case ncclUint64: RandomizeType<uint64_t>(ptr, count, seed); break;
    case ncclHalf:   RandomizeHalf          (ptr, count, seed); break;
    case ncclFloat:  RandomizeType<float>   (ptr, count, seed); break;
    case ncclDouble: RandomizeType<double>  (ptr, count, seed); break;
  }
}

template<typename T, int OP> __global__ static
void accumKern(T* acum, const T* contrib, size_t N) {
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int offset = blockDim.x*gridDim.x;
  for(int i=tid; i<N; i+=offset) {
    T c = contrib[i];
    T a = acum[i];
    if(OP == ncclSum) {
      acum[i] = a+c;
    } else if(OP == ncclProd) {
      acum[i] = a*c;
    } else if(OP == ncclMax) {
      acum[i] = (a > c) ? a : c;
    } else if(OP == ncclMin) {
      acum[i] = (a < c) ? a : c;
    }
  }
}

template<> __global__
void accumKern<half, ncclSum>(half* acum, const half* contrib, size_t N) {
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int offset = blockDim.x*gridDim.x;
  for(int i=tid; i<N; i+=offset) {
    float c = __half2float(contrib[i]);
    float a = __half2float(acum[i]);
    acum[i] = __float2half( a + c );
  }
}

template<> __global__
void accumKern<half, ncclProd>(half* acum, const half* contrib, size_t N) {
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int offset = blockDim.x*gridDim.x;
  for(int i=tid; i<N; i+=offset) {
    float c = __half2float(contrib[i]);
    float a = __half2float(acum[i]);
    acum[i] = __float2half( a * c );
  }
}

template<> __global__
void accumKern<half, ncclMax>(half* acum, const half* contrib, size_t N) {
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int offset = blockDim.x*gridDim.x;
  for(int i=tid; i<N; i+=offset) {
    float c = __half2float(contrib[i]);
    float a = __half2float(acum[i]);
    acum[i] = __float2half( (a>c) ? a : c );
  }
}

template<> __global__
void accumKern<half, ncclMin>(half* acum, const half* contrib, size_t N) {
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int offset = blockDim.x*gridDim.x;
  for(int i=tid; i<N; i+=offset) {
    float c = __half2float(contrib[i]);
    float a = __half2float(acum[i]);
    acum[i] = __float2half( (a<c) ? a : c );
  }
}

template<typename T>
void accVecType(void* out, void* in, size_t n, ncclRedOp_t op) {
  switch(op) {
    case ncclSum:  accumKern<T, ncclSum> <<<256,256>>>((T*)out, (T*)in, n); break;
    case ncclProd: accumKern<T, ncclProd><<<256,256>>>((T*)out, (T*)in, n); break;
    case ncclMax:  accumKern<T, ncclMax> <<<256,256>>>((T*)out, (T*)in, n); break;
    case ncclMin:  accumKern<T, ncclMin> <<<256,256>>>((T*)out, (T*)in, n); break;
    default:
      printf("Unknown reduction operation.\n");
      exit(EXIT_FAILURE);
  }
}

void Accumulate(void* out, void* in, size_t n, ncclDataType_t type, ncclRedOp_t op) {
  switch (type) {
    case ncclChar:   accVecType<int8_t>   (out, in, n, op); break;
#if NCCL_MAJOR >= 2
    case ncclUint8:  accVecType<uint8_t>  (out, in, n, op); break;
#endif
    case ncclInt:  accVecType<int32_t>  (out, in, n, op); break;
#if NCCL_MAJOR >= 2
    case ncclUint32: accVecType<uint32_t> (out, in, n, op); break;
#endif
    case ncclInt64:  accVecType<int64_t>  (out, in, n, op); break;
    case ncclUint64: accVecType<uint64_t> (out, in, n, op); break;
    case ncclHalf:   accVecType<half>     (out, in, n, op); break;
    case ncclFloat:  accVecType<float>    (out, in, n, op); break;
    case ncclDouble: accVecType<double>   (out, in, n, op); break;
    default:
      printf("Unknown reduction type.\n");
      exit(EXIT_FAILURE);
  }
}

void Barrier(struct threadArgs_t* args)
{
  while (args->barrier[args->barrier_idx] != args->thread) pthread_yield();

  args->barrier[args->barrier_idx] = args->thread + 1;

  if (args->thread+1 == args->nThreads) {
#ifdef MPI_SUPPORT
    MPI_Barrier(MPI_COMM_WORLD);
#endif
    args->barrier[args->barrier_idx] = 0;
  } else {
    while (args->barrier[args->barrier_idx]) pthread_yield();
  }

  args->barrier_idx=!args->barrier_idx;
}

void RandomizeAccumulate(void* data, void* accum, size_t count, ncclDataType_t type, ncclRedOp_t op, int seed, int rank) {
  Randomize(data, count, type, seed);
  if (rank == 0) {
    CUDACHECK(hipMemcpy(accum, data, count*wordSize(type), hipMemcpyDeviceToHost));
  } else {
    Accumulate(accum, data, count, type, op);
  }
}

double CheckData(struct threadArgs_t* args, ncclDataType_t type, ncclRedOp_t op, int root, int in_place) {
  size_t count = args->expectedBytes/wordSize(type);
  double maxDelta = 0.0;
  for (int i=0; i<args->nGpus; i++) {
    int device;
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    NCCLCHECK(ncclCommCuDevice(args->comms[i], &device));
    CUDACHECK(hipSetDevice(device));
    void *data = in_place ? ((void *)((uintptr_t)args->recvbuffs[i] + args->recvInplaceOffset*rank)) : args->recvbuffs[i];
    CheckDelta(data , args->expected[i], count, type, args->delta);
    hipDeviceSynchronize();
    maxDelta = std::max(*(args->deltaHost), maxDelta);

#ifdef DEBUG_PRINT
    if (rank == 0) {
       int *temp = (int *)malloc(args->expectedBytes);

       printf("\n Expected: ");
       for(int j=0; j<args->expectedBytes/sizeof(int); j++) {
       	printf("%d:%d ", j, *((int *)args->expectedHost[0] + j));
       }
       printf("\n");

       hipMemcpy(temp, data, args->expectedBytes, hipMemcpyDeviceToHost);
       printf("\n Actual: ");
       for (int j=0; j<args->expectedBytes/sizeof(int); j++) {
       	printf("%d:%d ", j, *((int *)temp + j));
       }
       printf("\n");
       free(temp);
    }
#endif
  }
  double nranks = args->nProcs*args->nThreads*args->nGpus;
  if (maxDelta > DeltaMaxValue(type)*(nranks - 1)) args->errors[0]++;
  return maxDelta;
}

void InitSend(struct threadArgs_t* args, ncclDataType_t type, ncclRedOp_t op, int root, int in_place, int is_first) {
  size_t count = args->sendBytes / wordSize(type);
  static int rep = 1;
  for (int i=0; i<args->nGpus; i++) {
    int device;
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    NCCLCHECK(ncclCommCuDevice(args->comms[i], &device));
    CUDACHECK(hipSetDevice(device));
    void* data = in_place ? (void *)((uintptr_t)args->recvbuffs[i] + args->sendInplaceOffset*rank) : args->sendbuffs[i];
    int seed = rank+count+rep+in_place;
    Randomize(data, count, type, seed);

#ifdef DEBUG_PRINT
    if (rank == 2) {
       int *temp = (int *)malloc(args->sendBytes);
       hipMemcpy(temp, data, args->sendBytes, hipMemcpyDeviceToHost);
       printf("\n Send Data at rank %d:", rank);
       for (int i=0; i<args->sendBytes/sizeof(int); i++) {
       	printf("%d:%d ", i, *((int *)temp + i));
       }
       printf("\n");
       free(temp);
    }
#endif

    hipDeviceSynchronize();
  }
  rep++;
}

#define CHECK 1

void startColl(struct threadArgs_t* args, ncclDataType_t type, ncclRedOp_t op, int root, int in_place, int thread_offset) {
  size_t count = args->nbytes / wordSize(type);

  if (swap_args) {
      args = (struct threadArgs_t*)args->proc_args + (args->thread + thread_offset)%args->nThreads;
  }

  if (args->nGpus == 1) {
    int rank = args->proc*args->nThreads + args->thread;
    RunColl((void*)(in_place ? ((void *)((uintptr_t)args->recvbuffs[0] + args->sendInplaceOffset*rank)) : args->sendbuffs[0]),
        (void*)(in_place ? (void*)((uintptr_t)args->recvbuffs[0] + args->recvInplaceOffset*rank) : args->recvbuffs[0]),
        count, type, op, root, args->comms[0], args->streams[0]);
  } else {
    NCCLCHECK(ncclGroupStart());
    for (int i = 0; i < args->nGpus; i++) {
#ifndef NCCL_MAJOR
      int cudaDev;
      NCCLCHECK(ncclCommCuDevice(args->comms[i], &cudaDev));
      CUDACHECK(hipSetDevice(cudaDev));
#endif
      int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
      RunColl((void*)(in_place ? ((void *)((uintptr_t)args->recvbuffs[i] + args->sendInplaceOffset*rank)) : args->sendbuffs[i]),
          (void*)(in_place ? (void*)((uintptr_t)args->recvbuffs[i] + args->recvInplaceOffset*rank) : args->recvbuffs[i]),
          count, type, op, root, args->comms[i], args->streams[i]);
    }
    NCCLCHECK(ncclGroupEnd());
  }

  if (swap_args || blocking_coll) {
    //if args have been swapped, complete op before returning
    for (int i = 0; i < args->nGpus; ++i) {
      hipError_t err = hipErrorNotReady;
      while (err == hipErrorNotReady) {
          err = hipStreamQuery(args->streams[i]);
          pthread_yield();
      }
      CUDACHECK(err);
    }
  }
  if (blocking_coll) Barrier(args);
}

void completeColl(struct threadArgs_t* args) {
  //it swap_args was enabled, op would have been completed immediately
  if (swap_args || blocking_coll) return;

  for (int i = 0; i < args->nGpus; ++i) {
    hipError_t err = hipErrorNotReady;
    while (err == hipErrorNotReady) {
        err = hipStreamQuery(args->streams[i]);
        pthread_yield();
    }
    CUDACHECK(err);
  }
}

void BenchTime(struct threadArgs_t* args, ncclDataType_t type, ncclRedOp_t op, int root, int in_place) {
  size_t count = args->nbytes / wordSize(type);

  // Sync
  startColl(args, type, op, root, in_place, 0);
  completeColl(args);

  Barrier(args);

  // Performance Benchmark
  auto start = std::chrono::high_resolution_clock::now();
  for (int iter = 0; iter < iters; iter++) {
      startColl(args, type, op, root, in_place, iter);
  }
  completeColl(args);

  auto delta = std::chrono::high_resolution_clock::now() - start;
  double deltaSec = std::chrono::duration_cast<std::chrono::duration<double>>(delta).count();
  deltaSec = deltaSec/iters;

  double algBw, busBw;
  GetBw(count, wordSize(type), deltaSec, &algBw, &busBw, args->nProcs*args->nThreads*args->nGpus);

  Barrier(args);

  if (datacheck) {
      InitSend(args, type, op, root, in_place, args->thread == 0 ? 1 : 0);
      InitRecvResult(args, type, op, root, in_place, args->thread == 0 ? 1 : 0);
      hipDeviceSynchronize();
  }

  //test validation in single itertion, should ideally be included into the multi-iteration run
  startColl(args, type, op, root, in_place, 0);
  completeColl(args);

  double maxDelta = 0;
#ifdef CHECK
  if (datacheck) {
     maxDelta = CheckData(args, type, op, root, in_place);
  } else {
     maxDelta = -1.0;
  }
#else
     maxDelta = -1.0;
#endif

  //aggregate delta from all threads and procs
  Barrier(args);
  if (args->thread == 0) {
      for (int i=1; i<args->nThreads; i++) {
          maxDelta += args->deltaThreads[i];
      }
#ifdef MPI_SUPPORT
      MPI_Allreduce(MPI_IN_PLACE, &maxDelta, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
#endif
  }
  Barrier(args);

  if (datacheck) {
     PRINT("  %7.3f  %5.2f  %5.2f  %7.0le", deltaSec * 1.0E3, algBw, busBw,
         maxDelta);
  } else {
     PRINT("  %7.3f  %5.2f  %5.2f  \tN/A", deltaSec * 1.0E3, algBw, busBw);
  }

  args->bw[0] += busBw;
  args->bw_count[0]++;
}

void setupArgs(size_t size, ncclDataType_t type, struct threadArgs_t* args) {
  int nranks = args->nProcs*args->nGpus*args->nThreads;
  size_t count, sendCount, recvCount, paramCount, sendInplaceOffset, recvInplaceOffset, procSharedCount;
  int sameExpected;

  count = size / wordSize(type);
  getCollByteCount(&sendCount, &recvCount, &paramCount, &sendInplaceOffset, &recvInplaceOffset, &procSharedCount, &sameExpected, (size_t)count, (size_t)nranks);

  args->nbytes = paramCount * wordSize(type);
  args->sendBytes = sendCount * wordSize(type);
  args->expectedBytes = recvCount * wordSize(type);
  args->sendInplaceOffset = sendInplaceOffset * wordSize(type);
  args->recvInplaceOffset = recvInplaceOffset * wordSize(type);
}

void TimeTest(struct threadArgs_t* args, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName, int root, int inPlace) {
  // Warm-up
  setupArgs(args->maxbytes, type, args);
  for (int iter = 0; iter < warmup_iters; iter++) {
     startColl(args, type, op, root, 0, iter);
  }
  completeColl(args);

  // Benchmark
  for (size_t size = args->minbytes; size<=args->maxbytes; size = ((args->stepfactor > 1) ? size*args->stepfactor : size+args->stepbytes)) {
      setupArgs(size, type, args);
      print_line_header(max(args->sendBytes, args->expectedBytes), args->nbytes / wordSize(type), typeName, opName, root);
      BenchTime(args, type, op, root, 0);
      if (inPlace) BenchTime(args, type, op, root, 1);
      PRINT("\n");
  }
}


void* threadRunTests(void* args) {
  struct threadArgs_t* targs = (struct threadArgs_t*)args;
  // Set device to the first of our GPUs. If we don't do that, some operations
  // will be done on the current GPU (by default : 0) and if the GPUs are in
  // exclusive mode those operations will fail.
  int gpuid = targs->localRank*targs->nThreads*targs->nGpus + targs->thread*targs->nGpus;
  CUDACHECK(hipSetDevice(gpuid));

  RunTest(targs, ncclroot, (ncclDataType_t)nccltype, test_typenames[nccltype], (ncclRedOp_t)ncclop, test_opnames[ncclop]);

  return NULL;
}

void* threadInit(void* args) {
  struct threadArgs_t* targs = (struct threadArgs_t*)args;
  char hostname[1024];
  getHostName(hostname, 1024);
  int nranks =  targs->nProcs*targs->nThreads*targs->nGpus;

  //set main thread again
  is_main_thread = (targs->proc == 0 && targs->thread == 0) ? 1 : 0;

  NCCLCHECK(ncclGroupStart());
  for (int i=0; i<targs->nGpus; i++) {
    int rank = targs->proc*targs->nThreads*targs->nGpus + targs->thread*targs->nGpus + i;
    int gpuid = targs->localRank*targs->nThreads*targs->nGpus + targs->thread*targs->nGpus + i;
    CUDACHECK(hipSetDevice(gpuid));
    NCCLCHECK(ncclCommInitRank(targs->comms+i, nranks, targs->ncclId, rank));
  }
  NCCLCHECK(ncclGroupEnd());

  PRINT("# Using devices\n");
  for (int p=0; p<targs->nProcs; p++) {
    if (p == targs->proc) {
      for (int t=0; t<targs->nThreads; t++) {
        if (t == targs->thread) {
          for (int i=0; i<targs->nGpus; i++) {
            int cudaDev;
            int rank;
            hipDeviceProp_t prop;
            NCCLCHECK(ncclCommCuDevice(targs->comms[i], &cudaDev));
            NCCLCHECK(ncclCommUserRank(targs->comms[i], &rank));
            CUDACHECK(hipGetDeviceProperties(&prop, cudaDev));
            printf("#   Rank %2d on %10s device %2d [0x%02x] %s\n", rank, hostname, cudaDev,
                prop.pciBusID, prop.name);
            fflush(stdout);
          }
          Barrier(targs);
          fflush(stdout);
	}
      }
    }
  }

  threadRunTests(args);

  return NULL;
}

void AllocateBuffs(void **sendbuff, size_t sendBytes, void **recvbuff, size_t recvBytes, void **expected, void **expectedHost, size_t nbytes, int nranks, int sameExpected) {
    static int is_first = 1;
    static void *cached_ptr = NULL;
    static void *cached_hostptr = NULL;

    CUDACHECK(hipMalloc(sendbuff, sendBytes));
    //work around for inline reduce scatter where recv count is smaller that send count
    CUDACHECK(hipMalloc(recvbuff, (sendBytes > recvBytes) ? sendBytes : recvBytes));

    if (is_first || !sameExpected) {
        *expectedHost = malloc(recvBytes);
        CUDACHECK(hipHostRegister(*expectedHost, recvBytes, hipHostRegisterPortable | hipHostRegisterMapped));
        CUDACHECK(hipHostGetDevicePointer(expected, *expectedHost, 0));
        cached_ptr = *expected;
        cached_hostptr = *expectedHost;
        is_first = 0;
    } else {
        *expected = cached_ptr;
        *expectedHost = cached_hostptr;
    }
}

int ncclstringtotype(char *str) {
    for (int t=0; t<ncclNumTypes; t++) {
      if (strcmp(str, test_typenames[t]) == 0) {
        return t;
      }
    }
    if (strcmp(str, "all") == 0) {
      return -1;
    }
    printf("invalid type %s, defaulting to %s .. \n", str, test_typenames[nccltype]);
    return nccltype;
}

int ncclstringtoop (char *str) {
    for (int o=0; o<ncclNumOps; o++) {
      if (strcmp(str, test_opnames[o]) == 0) {
        return o;
      }
    }
    if (strcmp(str, "all") == 0) {
      return -1;
    }
    printf("invalid op %s, defaulting to %s .. \n", str, test_opnames[ncclop]);
    return ncclop;
}

int main(int argc, char* argv[]) {
 int nThreads = 1, nGpus = 1;
 size_t minBytes = 32*1024*1024, maxBytes = 32*1024*1024, stepBytes = 1*1024*1024, stepFactor = 1;
 int longindex;
 int nProcs = 1, proc = 0;
 int localRank = 0;
 char hostname[1024];
 getHostName(hostname, 1024);

 static struct option longopts[] = {
    {"nthreads", required_argument, 0, 't'},
    {"ngpus", required_argument, 0, 'g'},
    {"minbytes", required_argument, 0, 'b'},
    {"maxbytes", required_argument, 0, 'e'},
    {"stepbytes", required_argument, 0, 'i'},
    {"stepfactor", required_argument, 0, 'f'},
    {"iters", required_argument, 0, 'n'},
    {"warmup_iters", required_argument, 0, 'w'},
    {"swap_comms", required_argument, 0, 's'},
    {"parallel_init", required_argument, 0, 'p'},
    {"check", required_argument, 0, 'c'},
    {"blocking", required_argument, 0, 'z'},
    {"op", required_argument, 0, 'o'},
    {"datatype", required_argument, 0, 'd'},
    {"root", required_argument, 0, 'r'},
    {"hostname", required_argument, 0, 'y'},
    {"help", no_argument, 0, 'h'}
 };

 while(1) {
      int c;
      c = getopt_long(argc, argv, "t:g:b:e:i:f:n:w:s:p:c:o:d:r:z:hy:", longopts, &longindex);

      if (c == -1)
         break;

      switch(c) {
         case 't':
             nThreads = strtol(optarg, NULL, 0);
             break;
         case 'g':
             nGpus = strtol(optarg, NULL, 0);
             break;
         case 'b':
             minBytes = (size_t)parsesize(optarg);
             break;
         case 'e':
             maxBytes = (size_t)parsesize(optarg);
             break;
         case 'i':
             stepBytes = strtol(optarg, NULL, 0);
             break;
         case 'f':
             stepFactor = strtol(optarg, NULL, 0);
             break;
	 case 'n':
	     iters = (int)strtol(optarg, NULL, 0);
	     break;
	 case 'w':
	     warmup_iters = (int)strtol(optarg, NULL, 0);
	     break;
	 case 's':
	     swap_args = (int)strtol(optarg, NULL, 0);
	     break;
	 case 'c':
	     datacheck = (int)strtol(optarg, NULL, 0);
	     break;
	 case 'p':
	     parallel_init = (int)strtol(optarg, NULL, 0);
	     break;
	 case 'o':
	     ncclop = ncclstringtoop(optarg);
	     break;
	 case 'd':
	     nccltype = ncclstringtotype(optarg);
	     break;
	 case 'r':
	     ncclroot = strtol(optarg, NULL, 0);
	     break;
     case 'y':
         ncclhost = optarg;
         break;
	 case 'z':
	     blocking_coll = strtol(optarg, NULL, 0);
	     break;
         case 'h':
	         printf("USAGE: ./test \n\t"
	 	 "[-t,--nthreads <num threads>] \n\t "
		 "[-g,--ngpus <gpus per thread>] \n\t "
		 "[-b,--minbytes <min size in bytes>] \n\t "
		 "[-e,--maxbytes <max size in bytes>] \n\t "
	         "[-i,--stepbytes <increment size>] \n\t "
		 "[-f,--stepfactor <increment factor>] \n\t "
		 "[-n,--iters <iteration count>] \n\t "
		 "[-w,--warmup_iters <warmup iteration count>] \n\t"
		 "[-s,--swap_args <0/1>] \n\t "
		 "[-p,--parallel_init <0/1>] \n\t "
		 "[-c,--check <0/1>] \n\t "
		 "[-o,--op <sum/prod/min/max/all>] \n\t "
		 "[-d,--datatype <nccltype/all>] \n\t "
		 "[-r,--root <root>] \n\t "
		 "[-z,--blocking <0/1>] \n\t "
		 "[-h,--help]\n");
	         return 0;
	 default:
	         printf("invalid option \n");
	         printf("USAGE: ./test \n\t"
	 	 "[-t,--nthreads <num threads>] \n\t "
		 "[-g,--ngpus <gpus per thread>] \n\t "
		 "[-b,--minbytes <min size in bytes>] \n\t "
		 "[-e,--maxbytes <max size in bytes>] \n\t "
	         "[-i,--stepbytes <increment size>] \n\t "
		 "[-f,--stepfactor <increment factor>] \n\t "
		 "[-n,--iters <iteration count>] \n\t "
		 "[-w,--warmup_iters <warmup iteration count>] \n\t"
		 "[-s,--swap_args <0/1>] \n\t "
		 "[-p,--parallel_init <0/1>] \n\t "
		 "[-c,--check <0/1>] \n\t "
		 "[-o,--op <sum/prod/min/max/all>] \n\t "
		 "[-d,--datatype <nccltype/all>] \n\t "
		 "[-r,--root <root>] \n\t "
		 "[-z,--blocking <0/1>] \n\t "
		 "[-h,--help]\n");
	         return 0;
      }
  }

  // Make sure everyline is flushed so that we see the progress of the test
  setlinebuf(stdout);

#ifdef MPI_SUPPORT
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &nProcs);
  MPI_Comm_rank(MPI_COMM_WORLD, &proc);
  uint64_t hostHashs[nProcs];
  if (ncclhost != NULL) {
      hostHashs[proc] = getHostHash(ncclhost);
  } else {
    hostHashs[proc] = getHostHash(hostname);
  }
  MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD);
  for (int p=0; p<nProcs; p++) {
    if (p == proc) break;
    if (hostHashs[p] == hostHashs[proc]) localRank++;
  }
#endif
  is_main_thread = (proc == 0) ? 1 : 0;

  if (proc == 0) {
      printf("nThread %d nGpus %d minBytes %ld maxBytes %ld step: %ld(%s) warmup iters: %d iters: %d validation: %d \n", nThreads, nGpus, minBytes, maxBytes,
      			(stepFactor > 1)?stepFactor:stepBytes, (stepFactor > 1)?"factor":"bytes", warmup_iters, iters, datacheck);
      if (swap_args) printf("Swap Comms Enabled: swapping communicators among threads for each iteration \n");
      if (blocking_coll) printf("Blocking Enabled: wait for completion and barrier after each collective \n");
      if (parallel_init) printf("Parallel Init Enabled: threads call into NcclInitRank concurrently \n");
  }

  ncclUniqueId ncclId;
  if (proc == 0) {
    NCCLCHECK(ncclGetUniqueId(&ncclId));
  }
#ifdef MPI_SUPPORT
  MPI_Bcast(&ncclId, sizeof(ncclId), MPI_BYTE, 0, MPI_COMM_WORLD);
#endif
  hipStream_t streams[nGpus*nThreads];
  void* sendbuffs[nGpus*nThreads];
  void* recvbuffs[nGpus*nThreads];
  void* expected[nGpus*nThreads];
  void* expectedHost[nGpus*nThreads];
  void *procSharedHost, *procShared;
  size_t sendBytes, recvBytes, paramBytes, procSharedBytes, sendInplaceOffset, recvInplaceOffset;
  int sameExpected;

  getCollByteCount(&sendBytes, &recvBytes, &paramBytes, &sendInplaceOffset, &recvInplaceOffset, &procSharedBytes, &sameExpected, (size_t)maxBytes, (size_t)nProcs*nGpus*nThreads);

  for (int i=0; i<nGpus*nThreads; i++) {
    CUDACHECK(hipSetDevice(localRank*nThreads*nGpus+i));
    AllocateBuffs(sendbuffs+i, sendBytes, recvbuffs+i, recvBytes, expected+i, expectedHost+i, (size_t)maxBytes, nProcs*nThreads*nGpus, sameExpected);
    CUDACHECK(hipStreamCreate(streams+i));
  }

  if (procSharedBytes > 0) {
      procSharedHost = malloc(procSharedBytes);
      CUDACHECK(hipHostRegister(procSharedHost, procSharedBytes, hipHostRegisterPortable | hipHostRegisterMapped));
      CUDACHECK(hipHostGetDevicePointer(&procShared, procSharedHost, 0));
  }

  //if parallel init is not selected, use main thread to initialize NCCL
  ncclComm_t* comms = (ncclComm_t*)malloc(sizeof(ncclComm_t)*nThreads*nGpus);
  if (!parallel_init) {
     if (nProcs == 1) {
       int gpuArray[nGpus*nThreads];
       for (int i=0; i<nGpus*nThreads; i++) gpuArray[i] = i;
       NCCLCHECK(ncclCommInitAll(comms, nGpus*nThreads, gpuArray));
     } else {
       NCCLCHECK(ncclGroupStart());
       for (int i=0; i<nGpus*nThreads; i++) {
         CUDACHECK(hipSetDevice(localRank*nThreads*nGpus+i));
         NCCLCHECK(ncclCommInitRank(comms+i, nProcs*nThreads*nGpus, ncclId, proc*nThreads*nGpus+i));
       }
       NCCLCHECK(ncclGroupEnd());
     }

     PRINT("# NCCL Tests compiled with NCCL %d.%d\n", NCCL_MAJOR, NCCL_MINOR);
     PRINT("# Using devices\n");
     for (int p=0; p<nProcs; p++) {
       if (p == proc) {
         for (int i=0; i<nThreads*nGpus; i++) {
           int cudaDev;
           int rank;
           hipDeviceProp_t prop;
           NCCLCHECK(ncclCommCuDevice(comms[i], &cudaDev));
           NCCLCHECK(ncclCommUserRank(comms[i], &rank));
           CUDACHECK(hipGetDeviceProperties(&prop, cudaDev));
           printf("#   Rank %2d on %10s device %2d [0x%02x] %s\n", rank, hostname, cudaDev,
               prop.pciBusID, prop.name);
           fflush(stdout);
         }
       }
#ifdef MPI_SUPPORT
       MPI_Barrier(MPI_COMM_WORLD);
#endif
       fflush(stdout);
     }
  }

  int errors[nThreads];
  double bw[nThreads];
  double delta[nThreads];
  int bw_count[nThreads];
  for (int t=0; t<nThreads; t++) {
    bw[t] = 0.0;
    errors[t] = bw_count[t] = 0;
  }

  PRINT("\n");
  print_header();

  int* sync = (int*)calloc(2, sizeof(int));
  int* barrier = (int*)calloc(2, sizeof(int));

  pthread_t threads[nThreads];
  struct threadArgs_t args[nThreads];

  for (int t=nThreads-1; t>=0; t--) {
    args[t].proc_args = (void *)args;
    args[t].minbytes=minBytes;
    args[t].maxbytes=maxBytes;
    args[t].stepbytes=stepBytes;
    args[t].stepfactor=stepFactor;
    args[t].localRank = localRank;

    args[t].nProcs=nProcs;
    args[t].proc=proc;
    args[t].nThreads=nThreads;
    args[t].thread=t;
    args[t].nGpus=nGpus;
    args[t].sendbuffs = sendbuffs+t*nGpus;
    args[t].recvbuffs = recvbuffs+t*nGpus;
    args[t].ncclId = ncclId;
    args[t].comms=comms+t*nGpus;
    args[t].streams=streams+t*nGpus;

    args[t].expectedHost = expectedHost + t*nGpus;
    args[t].expected = expected + t*nGpus;
    args[t].procSharedHost = procSharedHost;
    args[t].procShared = procShared;
    args[t].barrier = (volatile int*)barrier;
    args[t].barrier_idx = 0;
    args[t].sync = (volatile int*)sync;
    args[t].sync_idx = 0;
    args[t].deltaThreads = delta;
    args[t].deltaHost = (delta + t);
    CUDACHECK(hipHostRegister(args[t].deltaHost, sizeof(double), hipHostRegisterPortable|hipHostRegisterMapped));
    CUDACHECK(hipHostGetDevicePointer(&args[t].delta, args[t].deltaHost, 0));
    args[t].errors=errors+t;
    args[t].bw=bw+t;
    args[t].bw_count=bw_count+t;

    if (!parallel_init) {
       if (t)
         pthread_create(threads+t, NULL, threadRunTests, args+t);
       else
         threadRunTests(args);
    } else {
        if (t || (parallel_init && (proc == 0)))
         pthread_create(threads+t, NULL, threadInit, args+t);
       else
         threadInit(args);
    }
  }

  // Wait for other threads
  for (int t=nThreads-1; t>=0; t--) {
    if (t || (parallel_init && (proc == 0))) pthread_join(threads[t], NULL);
    errors[0] += errors[t];
    bw[0] += bw[t];
    bw_count[0] += bw_count[t];
  }

#ifdef MPI_SUPPORT
    MPI_Allreduce(MPI_IN_PLACE, &errors[0], 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
#endif

  for(int i=0; i<nGpus*nThreads; ++i)
    ncclCommDestroy(comms[i]);
  free(comms);

  char* str = getenv("NCCL_TESTS_MIN_BW");
  double check_avg_bw = str ? atof(str) : -1;
  bw[0] /= bw_count[0];

  PRINT(" Out of bounds values : %d %s\n", errors[0], errors[0] ? "FAILED" : "OK");
  PRINT(" Avg bus bandwidth    : %g %s\n", bw[0], check_avg_bw == -1 ? "" : (bw[0] < check_avg_bw*(0.9) ? "FAILED" : "OK"));
  PRINT("\n");
#ifdef MPI_SUPPORT
  MPI_Finalize();
#endif
  if (errors[0] || bw[0] < check_avg_bw*(0.9))
    exit(EXIT_FAILURE);
  else
    exit(EXIT_SUCCESS);
}
